#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 128

//*****************************************************************/
//function name:	corrcoefGPU
//arguments:		v[] (in) 2-D matrix, size * length				
//					size (in) - num of vectors 
//					length (in) - length of the vectors
//return value :	result of corrCoef - everyone by everyone
//*****************************************************************/
//kernel corrcoef function

__global__ void mean(double *dev_vec, duoble *dev_meanArray, int length)
{
	int block=blockIdx.x;
	int thread=threadIdx.x;
	int index = block*BLOCK_SIZE+thread; //voxel's num = line number
	double *pt=dev_vec+index*length;	//enter point to vec
	int i;
	double res=0;
	for (i=0;i<length;i++){
		res+=pt[i];
	}
	res/=legnth;
	dev_meanArray[index]=res;
	__syncthreads();
}

__global__ void corrCoefGPU(double *dev_vin, int i,double *dev_meanArray, double *dev_vout,int length)
{
	int block=blockIdx.x;
	int thread=threadIdx.x;
	int index = block*BLOCK_SIZE+thread;
	
	const double TINY=1.0e-20;
	
	double *current=dev_vin+i*length;	//enter point to vec, specific vector i
	double *pt=dev_vin+index*length;	//enter point to vec, for this one thread
	double syy=0.0,sxy=0.0,sxx=0.0;
	//TODO: parallel
	 for (int j=0;j<length;j++) 
	{     
        sxx += (current[j]-dev_meanArray[i])*(current[j]-dev_meanArray[i]);
        syy += (pt[j]-dev_meanArray[index])*(pt[j]-dev_meanArray[index]);
        sxy += (current[j]-dev_meanArray[i])*(pt[j]-dev_meanArray[index]);
    }

    dev_vout[index]=sxy/(sqrt(sxx*syy)+TINY);
}

void corrcoefGPU_kernel(double v[], int size, int length)
{
	double* dev_vin,dev_vout,dev_meanArray;
	hipMalloc ((void**)&dev_vin,size*length);
	hipMalloc ((void**)&dev_meanArray,size);
	hipMalloc ((void**)&dev_vout,size*size);
	
	hipMemcpy (dev_vin,v,size*length,hipMemcpyHostToDevice);
	dim3 dimBlock(BLOCK_SIZE,1);
	dim3 dimGrid();	
	
	mean<<<dimGrid,dimBlock>>>(dev_vin,dev_meanArray,length);
	
	for (int i=0;i<size;i++){
		corrCoefGPU<<<dimGrid,dimBlock>>>(dev_vin,i,dev_meanArray,dev_vout,length);
	}
	hipMemcpy (dev_vout,v,size*length,hipMemcpyHostToDevice);
}