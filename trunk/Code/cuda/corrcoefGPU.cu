#include "hip/hip_runtime.h"

#define BLOCK_SIZE 128

//*****************************************************************/
//function name:	corrcoefGPU
//arguments:		v[] (in) 2-D matrix, size * length				
//					size (in) - num of vectors 
//					length (in) - length of the vectors
//return value :	result of corrCoef - everyone by everyone
//*****************************************************************/
//kernel corrcoef function

void corrcoefGPU(double v[], int size, int length)
{
	double* dev_vin,dev_vout;
	hipMalloc ((void**)&dev_vin,size*length);
	hipMalloc ((void**)&dev_vout,size*size);
	
	hipMemcpy (dev_vin,v,size*length,hipMemcpyHostToDevice);
	for (int i=0;i<size;i++){
		
	}
}