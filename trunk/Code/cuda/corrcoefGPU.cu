#include "hip/hip_runtime.h"

#include"corrcoef.cuh"


//*****************************************************************/
//function name:	corrcoefGPU
//arguments:		v[] (in) 2-D matrix, size * length				
//					size (in) - num of vectors 
//					length (in) - length of the vectors
//return value :	result of corrCoef - everyone by everyone
//*****************************************************************/
//kernel corrcoef function

__global__ void mean(double *dev_vec, duoble *dev_meanArray, int length)
{
	int block=blockIdx.x;
	int thread=threadIdx.x;
	int index = block*BLOCK_SIZE+thread; //voxel's num = line number
	double *pt=dev_vec+index*length;	//enter point to vec
	int i;
	double res=0;
	for (i=0;i<length;i++){
		res+=pt[i];
	}
	res/=legnth;
	dev_meanArray[index]=res;
	__syncthreads();
}

__global__ void corrCoefGPU(double *dev_vin, int i,double *dev_meanArray, double *dev_vout,int length)
{
	int block=blockIdx.x;
	int thread=threadIdx.x;
	int index = block*BLOCK_SIZE+thread;
	
	const double TINY=1.0e-20;
	
	double *current=dev_vin+i*length;	//enter point to vec, specific vector i
	double *pt=dev_vin+index*length;	//enter point to vec, for this one thread
	double syy=0.0,sxy=0.0,sxx=0.0;
	
	 for (int j=0;j<length;j++) 
	{     
        sxx += (current[j]-dev_meanArray[i])*(current[j]-dev_meanArray[i]);
        syy += (pt[j]-dev_meanArray[index])*(pt[j]-dev_meanArray[index]);
        sxy += (current[j]-dev_meanArray[i])*(pt[j]-dev_meanArray[index]);
    }

    dev_vout[i][index]=sxy/(sqrt(sxx*syy)+TINY);
}

double* corrcoefGPU_kernel(double* v, double* res, int size, int length)
{
	double* dev_vin,dev_vout,dev_meanArray;
	int status=1;
	
	hipMalloc ((void**)&dev_vin,size*length);
	hipMalloc ((void**)&dev_meanArray,size);
	hipMalloc ((void**)&dev_vout,size*size);
	
	hipMemcpy (dev_vin,v,size*length,hipMemcpyHostToDevice);
	dim3 dimBlock(BLOCK_SIZE,1);
	dim3 dimGrid(size/BLOCK_SIZE,1);	
	
	mean<<<dimGrid,dimBlock>>>(dev_vin,dev_meanArray,length);
	
	for (int i=0;i<size;i++){
		corrCoefGPU<<<dimGrid,dimBlock>>>(dev_vin,i,dev_meanArray,dev_vout,length);
	}
	hipMemcpy (res,dev_vout,size*size,hipMemcpyHostToDevice);
	status=0;
	return status;
}